#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>
#include <vector>

#define D65_ROW 531		// D65�̍s��
#define D65_COL 2		// D65�̗�
#define OBS_ROW 441		// �W���ϑ��҂̍s��
#define OBS_COL 4		// �W���ϑ��҂̗�
#define DATA_ROW 391	// �v�Z�Ŏg�p����f�[�^�̍s�� (390 - 780 nm)
#define DATA_MIN 390	// �g�p������g���̍ŏ��l
#define DATA_MAX 780	// �g�p������g���̍ő�l
#define PI 3.141592		// �~����

#define BLOCKSIZE 371		// 1�u���b�N������̃X���b�h��
#define DATANUM 50			// �v�Z���鐔
#define CALCNUM 100		// �ׂ��悷�鐔
#define SIMNUM 1023			// �V�~�����[�V���������
#define LOOPNUM 10			// SIMNUM��̃V�~�����[�V�����J��Ԃ���

using namespace std;

/* CUDA�G���[�`�F�b�N */
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/* �t�@�C������f�[�^��ǂݍ��ފ֐� */
int getFileData(vector<vector<double> >& d65_data, vector<vector<double> >& obs_data) {
	/* �t�@�C���|�C���^ */
	FILE* fp_d65, * fp_obs;
	/* EOF�����o����ϐ� */
	int ret;
	/* �J�E���^�[ */
	int count = 0;

	/* D65�̓ǂݍ��� */
	/* �t�@�C���I�[�v�� */
	fp_d65 = fopen("./d65.csv", "r");
	/* �������J���Ă��邩���`�F�b�N */
	if (fp_d65 == NULL) {
		cout << "File open error" << endl;
		return -1;
	}

	/* �t�@�C���ǂݍ��� */
	for (int i = 0; i < D65_ROW; i++) {
		/* 1�s���ǂݍ��� */
		ret = fscanf(fp_d65, "%lf, %lf", &(d65_data[count][0]), &(d65_data[count][1]));
		/* �I������ */
		if (d65_data[count][0] == DATA_MAX) {
			count = 0;
			break;
		}
		/* �J�E���^�̍X�V */
		if (d65_data[count][0] >= DATA_MIN) {
			count++;
		}
		/* �G���[�����o�����ۂ̏��� */
		if (ret == EOF) {
			cout << "error" << endl;
			return -1;
		}
	}
	fclose(fp_d65);


	/* �W���ϑ��҂̓ǂݍ��� */
	/* �t�@�C���I�[�v�� */
	fp_obs = fopen("./std_obs_10deg.csv", "r");
	/* �������J���Ă��邩���`�F�b�N */
	if (fp_obs == NULL) {
		cout << "File open error" << endl;
		return -1;
	}

	/* �t�@�C���ǂݍ��� */
	for (int i = 0; i < OBS_ROW; i++) {
		/* 1�s���ǂݍ��� */
		ret = fscanf(fp_obs, "%lf, %lf, %lf, %lf", &(obs_data[i][0]), &(obs_data[i][1]), &(obs_data[i][2]), &(obs_data[i][3]));
		/* �I������ */
		if (obs_data[count][0] == DATA_MAX) {
			count = 0;
			break;
		}
		/* �J�E���^�̍X�V */
		if (obs_data[count][0] >= DATA_MIN) {
			count++;
		}
		/* �G���[�����o�����ۂ̏��� */
		if (ret == EOF) {
			cout << "error" << endl;
			return -1;
		}
	}
	fclose(fp_d65);

	return 0;
}

/* �K�E�V�A���̃V�t�g���v�Z����֐� */
void makeGaussShift(vector<vector<double> >& shift_data) {
	double mu = 0;			// �v�Z�Ŏg�p����~���[
	double sigma = 0;		// �v�Z�Ŏg�p����V�O�}
	double d_max = 0;		// ���������K�E�V�A���̒��̍ő�l
	double w_length = 0;	// �U����0-1�̊ԂŃ����_���ɂ��邽�߂Ɏg�p����

	/* �����̃V�[�h���� */
	srand((unsigned int)time(NULL));

	/* �g�`��10�p�^�[����������̂�10��Ń��[�v���� */
	for (int i = 0; i < 10; i++) {
		mu = (double)DATA_MIN + ((double)DATA_MAX - (double)DATA_MIN) / 10 * i;
		sigma = 5 + (95 * (double)rand() / RAND_MAX);

		/* �f�[�^�������v�Z���� */
		for (int j = 0; j < DATA_ROW; j++) {
			shift_data[j][i] = 1 / (sqrt(2 * PI) * sigma) * exp(-pow(((double)(DATA_MIN + j) - mu), 2) / (2 * sigma * sigma));
			/* �ő�l��ϐ��Ɋi�[����(�X�V����) */
			if (d_max < shift_data[j][i]) {
				d_max = shift_data[j][i];
			}
		}

		/* ���������K�E�V�A���𐳋K�����A�U����0-1�̊ԂŃ����_���ɂ��� */
		w_length = (double)rand() / RAND_MAX;	// 0-1�̊Ԃŗ�������
		for (int j = 0; j < DATA_ROW; j++) {
			shift_data[j][i] = shift_data[j][i] / d_max * w_length;
		}
		/* �ő�l������ */
		d_max = 0;
	}
}

/* vector�^����z��փf�[�^���R�s�[����֐� */
void cpyVecToArray(vector<vector<double> >& d65_data,
vector<vector<double> >& obs_data,
vector<vector<double> >& shift_data,
double* d65, double* obs_x, double* obs_y, double* obs_z, double* gauss_data) {
	for (int i = 0; i < DATA_ROW; i++) {
		d65[i] = d65_data[i][1];
		obs_x[i] = obs_data[i][1];
		obs_y[i] = obs_data[i][2];
		obs_z[i] = obs_data[i][3];
		for (int j = 0; j < 10; j++) {
			int aPos = DATA_ROW * j + i;
			gauss_data[aPos] = shift_data[i][j];
		}
	}
}

/* ���a�v�Z�̎��Ɏg�p����ϐ����v�Z */
int getRemain(void) {
	/* �]�� */
	int remain = 0;

	/* �]��v�Z */
	for (int i = 1; i < BLOCKSIZE; i *= 2) {
		remain = BLOCKSIZE - i;
	}

	/* �]��o�� */
	return remain;
}

/* �ϕ��v�Z�J�[�l�� */
template<int BLOCK_SIZE> __global__ void colorSim(double simNum,double *g_data,double *d65,double *obs_x,double *obs_y,double *obs_z,double *result,int remain) {
	/* CUDA�A�N�Z�X�p�ϐ� */
	int ix = threadIdx.x;
	int aPos = 0;
	/* �ǂ̃K�E�V�A�������߂邽�߂̕ϐ� */
	__shared__ int sim_order[10];
	/* �K�E�V�A���g�ݍ��킹�̔ԍ� */
	__shared__ double sim_num;
	/* ���ʂ��i�[����V�F�A�[�h������ */
	__shared__ double calc_data[BLOCK_SIZE][3];
	/* �������킹���K�E�V�A���̍ő�l */
	__shared__ double g_max;
	g_max = 0;
	/* �������킹���K�E�V�A�����i�[���� */
	double gaussian = 0;
	/* �������킹���K�E�V�A�����i�[(�ő�l��r�p) */
	__shared__ double g_comp[BLOCK_SIZE];
	/* ��r�p�V�F�A�[�h������������ */
	g_comp[ix] = 0;

	/* sim_order�w�l������ */
	if (ix == 0) {
		sim_num = blockIdx.x + simNum;
		int count = 512;	// �J�E���^
		for (int i = 0; i < 10; i++) {
			if (sim_num >= count) {
				sim_num -= count;
				sim_order[i] = 1;
			}
			else { 
				sim_order[i] = 0;
			}
			count = count / 2;
		}
		/*printf("%d %d %d %d %d %d %d %d %d %d\n", 
			sim_order[0], sim_order[1], sim_order[2], sim_order[3], sim_order[4],
			sim_order[5], sim_order[6], sim_order[7], sim_order[8], sim_order[9] );*/
	}

	/* �u���b�N���̃X���b�h���� */
	__syncthreads();

	/* �K�E�V�A���𑫂����킹�� */
	for (int i = 0; i < 10; i++) {
		aPos = i * BLOCK_SIZE + ix;
		if (sim_order[i] == 1) {
			gaussian += g_data[aPos];
			g_comp[ix] += g_data[aPos];
		}
	}

	/* �u���b�N���̃X���b�h���� */
	__syncthreads();

	/* �������킹���K�E�V�A���̍ő�l�����߂� */
	if (ix == 0) {
		for (int i = 0; i < BLOCK_SIZE; i++) {
			if (g_max < g_comp[i]) {
				g_max = g_comp[i];
			}
		}
	}

	/* �u���b�N���̃X���b�h���� */
	__syncthreads();

	/* g_max ��1�ȏ�̏ꍇ�A�ő�l��0.99�ɂȂ�悤�ɐ��K�� */
	if (g_max >= 1) {
		gaussian = gaussian / g_max * 0.99;
	}

	/* �u���b�N���̃X���b�h���� */
	__syncthreads();

	for (int i = 0; i < CALCNUM; i++) {
		/* �V�F�A�[�h�������Ƀf�[�^�i�[ */
		calc_data[ix][0] = d65[ix] * obs_x[ix] * pow(gaussian, (0.01 * i));
		calc_data[ix][1] = d65[ix] * obs_y[ix] * pow(gaussian, (0.01 * i));
		calc_data[ix][2] = d65[ix] * obs_z[ix] * pow(gaussian, (0.01 * i));

		/* �u���b�N���� */
		__syncthreads();

		/* �u���b�N���ƂɃ��_�N�V��������(���a�v�Z) */
		/* �]�肪0�o�Ȃ��ꍇ */
		if (remain != 0) {
			/* �]�����v�f�̃V�F�A�[�h�����������Z���� */
			if (ix < remain) {
				calc_data[ix][0] += calc_data[BLOCK_SIZE - ix - 1][0];
				calc_data[ix][1] += calc_data[BLOCK_SIZE - ix - 1][1];
				calc_data[ix][2] += calc_data[BLOCK_SIZE - ix - 1][2];
			}
		}

		/* ���a�v�Z���� */
		if (BLOCK_SIZE >= 256) { if (ix < 128) { calc_data[ix][0] += calc_data[ix + 128][0];
												 calc_data[ix][1] += calc_data[ix + 128][1];
												 calc_data[ix][2] += calc_data[ix + 128][2];
												}__syncthreads(); }
		if (BLOCK_SIZE >= 128) { if (ix < 64) { calc_data[ix][0] += calc_data[ix + 64][0];
												calc_data[ix][1] += calc_data[ix + 64][1];
												calc_data[ix][2] += calc_data[ix + 64][2];
												}__syncthreads(); }
		if (BLOCK_SIZE >= 64) { if (ix < 32) { calc_data[ix][0] += calc_data[ix + 32][0];
											   calc_data[ix][1] += calc_data[ix + 32][1];
											   calc_data[ix][2] += calc_data[ix + 32][2];
											 } __syncthreads();}
		if (BLOCK_SIZE >= 32) { if (ix < 16) { calc_data[ix][0] += calc_data[ix + 16][0];
											   calc_data[ix][1] += calc_data[ix + 16][1];
											   calc_data[ix][2] += calc_data[ix + 16][2];
											 } __syncthreads();
		}
		if (BLOCK_SIZE >= 16) { if (ix < 8) { calc_data[ix][0] += calc_data[ix + 8][0];
											  calc_data[ix][1] += calc_data[ix + 8][1];
											  calc_data[ix][2] += calc_data[ix + 8][2];
											}__syncthreads();
		}
		if (BLOCK_SIZE >= 8) { if (ix < 4) { calc_data[ix][0] += calc_data[ix + 4][0];
											 calc_data[ix][1] += calc_data[ix + 4][1];
											 calc_data[ix][2] += calc_data[ix + 4][2];
											} __syncthreads();
		}
		if (BLOCK_SIZE >= 4) { if (ix < 2) { calc_data[ix][0] += calc_data[ix + 2][0];
											 calc_data[ix][1] += calc_data[ix + 2][1];
											 calc_data[ix][2] += calc_data[ix + 2][2];
											} __syncthreads();
		}
		if (BLOCK_SIZE >= 2) { if (ix < 1) { calc_data[ix][0] += calc_data[ix + 1][0];
											 calc_data[ix][1] += calc_data[ix + 1][1];
											 calc_data[ix][2] += calc_data[ix + 1][2];
											} __syncthreads();
		}

		/*if (ix == 0) {
			for (int j = 1; j < BLOCK_SIZE; j++) {
				calc_data[ix][0] += calc_data[i][0];
				calc_data[ix][1] += calc_data[i][1];
				calc_data[ix][2] += calc_data[i][2];
			}
		}*/
		/* �l�o�� */
		if (ix == 0) {
			/* aPos�X�V */
			aPos = blockIdx.x * 3 * CALCNUM + i;
			//printf("%d %d\n", blockIdx.x,calc_data[ix]);
			result[aPos] = calc_data[0][0];

			/* aPos�X�V */
			aPos = blockIdx.x * 3 * CALCNUM + i + CALCNUM;
			//printf("%d %d\n", blockIdx.x,calc_data[ix]);
			result[aPos] = calc_data[0][1];

			/* aPos�X�V */
			aPos = blockIdx.x * 3 * CALCNUM + i + (2 * CALCNUM);
			//printf("%d %d\n", blockIdx.x,calc_data[ix]);
			result[aPos] = calc_data[0][2];

			//printf("%.3lf %.3lf %.3lf\n", calc_data[0][0], calc_data[0][1], calc_data[0][2]);
		}

		/* �u���b�N���� */
		__syncthreads();
	}
}

int main(void) {
	/* D65�̃f�[�^���i�[����z�� */
	vector<vector<double> > d65_data(DATA_ROW, vector<double>(D65_COL, 0));
	/*�W���ϑ��҂̃f�[�^���i�[����z�� */
	vector<vector<double> > obs_data(DATA_ROW, vector<double>(OBS_COL, 0));
	/*�K�E�V�A����10�i�[����z�� */
	vector<vector<double> > gauss_shift(DATA_ROW, vector<double>(10, 0));

	/* �]��v�Z */
	int remain = getRemain();

	/* �f�[�^������P�����z�� */
	double* d65, * obs_x, * obs_y, * obs_z, * gauss_data, * result, * fin_result;
	d65 = new double[DATA_ROW];
	obs_x= new double[DATA_ROW];
	obs_y = new double[DATA_ROW];
	obs_z = new double[DATA_ROW];
	gauss_data = new double[DATA_ROW * 10];
	result = new double[3 * DATANUM * CALCNUM];
	fin_result = new double[3 * SIMNUM * CALCNUM * LOOPNUM];

	/* CUDA�p�̕ϐ� */
	double* d_d65, * d_obs_x, * d_obs_y, * d_obs_z, * d_gauss_data, *d_result;
	char* d_sim_order;

	/* GPU�������m�� */
	hipMalloc((void**)&d_d65, DATA_ROW * sizeof(double));
	hipMalloc((void**)&d_obs_x, DATA_ROW * sizeof(double));
	hipMalloc((void**)&d_obs_y, DATA_ROW * sizeof(double));
	hipMalloc((void**)&d_obs_z, DATA_ROW * sizeof(double));
	hipMalloc((void**)&d_gauss_data, DATA_ROW * 10 * sizeof(double));
	hipMalloc((void**)&d_result, 3 * DATANUM * CALCNUM * sizeof(double));

	/* �t�@�C���ǂݍ��݊֐����s */
	int f_result = getFileData(d65_data, obs_data);

	/* vector��1�����z��֕ϊ� */
	cpyVecToArray(d65_data, obs_data, gauss_shift, d65, obs_x, obs_y, obs_z, gauss_data);

	/* CUDA�ւ̃������R�s�[ */
	hipMemcpy(d_d65, d65, DATA_ROW * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_obs_x, obs_x, DATA_ROW * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_obs_y, obs_y, DATA_ROW * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_obs_z, obs_z, DATA_ROW * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_gauss_data, gauss_data, DATA_ROW * 10 * sizeof(double), hipMemcpyHostToDevice);


	for (int i = 0; i < LOOPNUM; i++) {
		/* �K�E�V�A���v�Z */
		makeGaussShift(gauss_shift);
		/* vector��1�����z��֕ϊ� */
		cpyVecToArray(d65_data, obs_data, gauss_shift, d65, obs_x, obs_y, obs_z, gauss_data);
		/* CUDA�ւ̃������R�s�[ */
		hipMemcpy(d_gauss_data, gauss_data, DATA_ROW * 10 * sizeof(double), hipMemcpyHostToDevice);

		for(int j = 0; j < (SIMNUM - DATANUM); j += DATANUM) {
			colorSim<DATA_ROW> << <DATANUM, DATA_ROW >> > ((j+1), d_gauss_data, d_d65, d_obs_x, d_obs_y, d_obs_z, d_result, remain);
			hipDeviceSynchronize();

			/* ���ʂ̃R�s�[ */
			hipMemcpy(result, d_result, 3 * DATANUM * CALCNUM * sizeof(double), hipMemcpyDeviceToHost);

			for (int k = 0; k < (3 * DATANUM * CALCNUM); k++) {
				int aPos = (i * 3 * CALCNUM * SIMNUM) + (3 * CALCNUM * j) + k;
				fin_result[aPos] = result[k];
			}
		}

		/* ���[�v�ŗ]�����c��̐����V�~�����[�V���� */
		int r_num = SIMNUM % DATANUM - 1;
		int sim_num = SIMNUM - r_num - 1;
		colorSim<DATA_ROW> << <r_num, DATA_ROW >> > ((sim_num + 1), d_gauss_data, d_d65, d_obs_x, d_obs_y, d_obs_z, d_result, remain);

		/* ���ʂ̃R�s�[ */
		hipMemcpy(result, d_result, 3 * DATANUM * CALCNUM * sizeof(double), hipMemcpyDeviceToHost);

		for (int k = 0; k < (3 * r_num * CALCNUM); k++) { 
			int aPos = (i * 3 * CALCNUM * SIMNUM) + (3 * CALCNUM * sim_num) + k;
			fin_result[aPos] = result[k];
		}
	}

	/* ���ʂ��I�������𖞂����Ă���Ƃ��ɒl��0�ɂ��� */
	for (int i = 0; i < LOOPNUM; i++) {
		for (int j = 0; j < SIMNUM; j++) {
			for (int k = 0; k < CALCNUM; k++) {
				int aPos = (i * 3 * SIMNUM * CALCNUM) + (j * 3 * CALCNUM) + k;
				if ((fin_result[0] * 0.005) > fin_result[aPos] &&
					(fin_result[CALCNUM] * 0.005) > fin_result[aPos + CALCNUM] && 
					(fin_result[CALCNUM * 2] * 0.005) > fin_result[aPos + (CALCNUM * 2)]) {
					fin_result[aPos] = 0;
					fin_result[aPos + CALCNUM] = 0;
					fin_result[aPos + (CALCNUM * 2)] = 0;
				}
			}
		}
	}

	/* �o�̓f�B���N�g�� */
	string directory = "C:/Users/KoidaLab-WorkStation/Desktop/isomura_ws/color_simulation_result/sim_1023_10000_10/";

	/* �o�͂����t�@�C���̏����L�^����t�@�C�� */
	string f_info = "sim_file_info.txt";
	f_info = directory + f_info;
	ofstream o_f_info(f_info);

	/* �t�@�C���������� */
	for (int i = 0; i < LOOPNUM; i++) {
		/* �o�̓t�@�C���� */
		string fname1 = "sim_result_L_xyz_1023_";
		string fname2 = "sim_result_S_xyz_1023_";
		string fend = ".csv";
		fname1 = directory + fname1 + to_string(i + 1) + fend;
		fname2 = directory + fname2 + to_string(i + 1) + fend;

		/* �t�@�C���o�̓X�g���[�� */
		ofstream o_file1(fname1);
		ofstream o_file2(fname2);

		/* �o�͂����t�@�C���̏����L�^����t�@�C���Ƀt�@�C�������o�� */
		o_f_info << fname1 << endl;
		o_f_info << fname2 << endl;

		/* �t�@�C���ւ̏o�͌����w�� */
		o_file1 << fixed << setprecision(3);
		o_file2 << fixed << setprecision(3);
		for (int j = 0; j < CALCNUM; j++) {
			for (int k = 0; k < (SIMNUM - 1); k++) {
				int apos = j + ((3 * k) * CALCNUM) + (3 * SIMNUM * CALCNUM * i);

				double X = fin_result[apos];
				double Y = fin_result[apos + CALCNUM];
				double Z = fin_result[apos + (2 * CALCNUM)];

				/* XYZ == 0�̂Ƃ� */
				if (X == 0 && Y == 0 && Z == 0) {
					o_file1 << ",,,";
					o_file2 << ",,,";
				}

				/* ����ȊO�̂Ƃ� */
				else {
					double x = X / (X + Y + Z);
					double y = Y / (X + Y + Z);
					double z = Z / (X + Y + Z);

					o_file1 << X << "," << Y << "," << Z << ",";
					o_file2 << x << "," << y << "," << z << ",";
				}
			}
			int apos = j + (3 * (SIMNUM - 1)) * CALCNUM + (3 * SIMNUM * CALCNUM * i);

			double X = fin_result[apos];
			double Y = fin_result[apos + CALCNUM];
			double Z = fin_result[apos + (2 * CALCNUM)];

			/* XYZ == 0�̂Ƃ� */
			if (X == 0 && Y == 0 && Z == 0) {
				o_file1 << ",,";
				o_file2 << ",,";
			}

			/* ����ȊO�̂Ƃ� */
			else {
				double x = X / (X + Y + Z);
				double y = Y / (X + Y + Z);
				double z = Z / (X + Y + Z);

				o_file1 << X << "," << Y << "," << Z;
				o_file2 << x << "," << y << "," << z;
			}

			o_file1 << endl << flush;
			o_file2 << endl << flush;
		}
		/* �t�@�C���N���[�Y */
		o_file1.close();
		o_file2.close();
	}

	/* �f�o�C�X��������� */
	hipFree(d_d65);
	hipFree(d_gauss_data);
	hipFree(d_obs_x);
	hipFree(d_obs_y);
	hipFree(d_obs_z);
	hipFree(d_result); 

	/* �z�X�g��������� */
	delete[] d65;
	delete[] obs_x;
	delete[] obs_y;
	delete[] obs_z;
	delete[] gauss_data;
	delete[] result;
	delete[] fin_result;

	return 0;
}